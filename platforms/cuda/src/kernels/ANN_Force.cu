#include "hip/hip_runtime.h"
// real3 force1 = make_real3(- FORCE_CONSTANT[0] * (pos1.x - POTENTIAL_CENTER[0]), 0.0, 0.0) ; 
// real3 force2 = make_real3(0.0, 0.0, 0.0);

int num_of_rows, num_of_cols;

// forward propagation
// input layer
for (int ii = 0; ii < NUM_OF_NODES[0]; ii ++) {
    OUTPUT_0[ii] = INPUT_0[ii];
}
// layer 1
num_of_rows = NUM_OF_NODES[1];
num_of_cols = NUM_OF_NODES[0];
for (int ii = 0; ii < num_of_rows; ii ++) {
    INPUT_1[ii] = BIAS_0[ii];
    for (int jj = 0; jj < num_of_cols; jj ++) {
        INPUT_1[ii] += COEFF_0[ii * num_of_cols + jj] * OUTPUT_0[jj];
    }
}
if (LAYER_TYPES[0] == 0) { // linear
    for (int ii = 0; ii < num_of_rows; ii ++) {
        OUTPUT_1[ii] = INPUT_1[ii];
    }
}
else if (LAYER_TYPES[0] == 1) { // tanh
    for (int ii = 0; ii < num_of_rows; ii ++) {
        OUTPUT_1[ii] = tanh(INPUT_1[ii]);
    }
}
// layer 2
num_of_rows = NUM_OF_NODES[2];
num_of_cols = NUM_OF_NODES[1];
for (int ii = 0; ii < num_of_rows; ii ++) {
    INPUT_2[ii] = BIAS_1[ii];
    for (int jj = 0; jj < num_of_cols; jj ++) {
        INPUT_2[ii] += COEFF_1[ii * num_of_cols + jj] * OUTPUT_1[jj];
    }
}
if (LAYER_TYPES[0] == 0) { // linear
    for (int ii = 0; ii < num_of_rows; ii ++) {
        OUTPUT_2[ii] = INPUT_2[ii];
    }
}
else if (LAYER_TYPES[0] == 1) { // tanh
    for (int ii = 0; ii < num_of_rows; ii ++) {
        OUTPUT_2[ii] = tanh(INPUT_2[ii]);
    }
}

// backward propagation, INPUT_{0,1,2} are reused to store derivatives in each layer
// layer 2
for (int ii = 0; ii < NUM_OF_NODES[2]; ii ++) {
    INPUT_2[ii] = (OUTPUT_2[ii] - POTENTIAL_CENTER[ii]) * FORCE_CONSTANT[0];
}
if (LAYER_TYPES[1] == 1) {
    for (int ii = 0; ii < NUM_OF_NODES[2]; ii ++) {
        INPUT_2[ii] *= (1 - OUTPUT_2[ii] * OUTPUT_2[ii]);    
    }
}

// layer 1
num_of_rows = NUM_OF_NODES[2];
num_of_cols = NUM_OF_NODES[1];
for (int ii = 0; ii < num_of_cols; ii ++) {
    INPUT_1[ii] = 0;
    for (int jj = 0; jj < num_of_rows; jj ++) {
        INPUT_1[ii] += COEFF_1[ii + jj * num_of_cols] * INPUT_2[jj];
    }
}
if (LAYER_TYPES[1] == 1) {
    for (int ii = 0; ii < NUM_OF_NODES[1]; ii ++) {
        INPUT_1[ii] *= (1 - OUTPUT_1[ii] * OUTPUT_1[ii]);    
    }
}

// input layer
num_of_rows = NUM_OF_NODES[1];
num_of_cols = NUM_OF_NODES[0];
for (int ii = 0; ii < num_of_cols; ii ++) {
    INPUT_0[ii] = 0;
    for (int jj = 0; jj < num_of_rows; jj ++) {
        INPUT_0[ii] += COEFF_0[ii + jj * num_of_cols] * INPUT_1[jj];
    }
}
if (LAYER_TYPES[1] == 1) {
    for (int ii = 0; ii < NUM_OF_NODES[0]; ii ++) {
        INPUT_0[ii] *= (1 - OUTPUT_0[ii] * OUTPUT_0[ii]);    
    }
}

